#include "hip/hip_runtime.h"
//
// Created by Rahul Chunduru on 12/22.
//

#include "pipeline/pipeline_gpu.h"
#include "pipeline/gpu_kernel.h"

__global__
void  gpuCompute(void* pipeline_, int gpu_id_, void* batch) {
    // ((PipelineGPU*)pipeline_)->model_->device_models_[gpu_id_].get()->train_batch(*((shared_ptr<Batch>*)batch), ((PipelineGPU *)pipeline_)->pipeline_options_->gpu_model_average);
}

__global__
void  transferBatchToDevice(void* pipeline_, void* batch, int queue_choice) {
    *((shared_ptr<Batch>*)batch)->to((PipelineGPU*)pipeline_->model_->device_models_[queue_choice]->device_)

}

void launchKernel(void* pipeline_, int gpu_id_, void* batch) {
    gpuCompute<<<1, 64>>>(pipeline_, batch, queue_choice);
}

void transferDataToDevice(void* pipeline_, void* batch, int queue_choice) {
    transferBatchToDevice<<<1, 64>>>(pipeline_, batch, queue_choice);
}
