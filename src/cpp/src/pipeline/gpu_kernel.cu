#include "hip/hip_runtime.h"
//
// Created by Rahul Chunduru on 12/22.
//

#include "pipeline/pipeline_gpu.h"
#include "gpu_kernel.h"

__global__
void  gpuCompute(Pipeline pipeline_, int gpu_id_, shared_ptr<Batch> batch) {
    pipeline_->model_->device_models_[gpu_id_].get()->train_batch(batch, ((PipelineGPU *)pipeline_)->pipeline_options_->gpu_model_average);
}

void launchKernel(void* pipeline_, int gpu_id_, void* batch) {
    gpuCompute<<<1, 64>>>((Pipeline*)pipeline_, gpu_id_, (shared_ptr<Batch>)batch);
}
