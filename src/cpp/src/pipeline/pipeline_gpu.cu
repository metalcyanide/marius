#include "hip/hip_runtime.h"
//
// Created by Rahul Chunduru on 12/22.
//

#include "pipeline/pipeline_gpu.h"

__global__
void  gpuCompute(Pipeline pipeline_, int gpu_id_, shared_ptr<Batch> batch) {
    pipeline_->model_->device_models_[gpu_id_].get()->train_batch(batch, ((PipelineGPU *)pipeline_)->pipeline_options_->gpu_model_average);
}

void ComputeWorkerGPU::launchKernel() {
    gpuCompute<<<1, 64>>>(pipeline_, gpu_id_, batch);
}
